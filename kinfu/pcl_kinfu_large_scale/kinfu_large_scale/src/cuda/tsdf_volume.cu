#include "hip/hip_runtime.h"
/*
 * Software License Agreement (BSD License)
 *
 *  Point Cloud Library (PCL) - www.pointclouds.org
 *  Copyright (c) 2011, Willow Garage, Inc.
 *
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include "device.hpp"
//#include <boost/graph/buffer_concepts.hpp>

namespace pcl
{
  namespace device
  {
    namespace kinfuLS
    {
      template<typename T>
      __global__ void
      initializeVolume (int3 voxels_size,PtrStep<T> volume)
      {
        int x = threadIdx.x + blockIdx.x * blockDim.x;
        int y = threadIdx.y + blockIdx.y * blockDim.y;
        
        
        if (x < voxels_size.x && y < voxels_size.y)
        {
            T *pos = volume.ptr(y) + x;
            int z_step = voxels_size.y * volume.step / sizeof(*pos);

  #pragma unroll
            for(int z = 0; z < voxels_size.z; ++z, pos+=z_step)
              pack_tsdf (0.f, 0, *pos);
        }
      }

      template<typename T>
      __global__ void
      clearSphereKernel(PtrStep<T> volume,int3 volume_size,int3 shift,float3 center,float radius,bool set_to_empty)
      {
        int x = threadIdx.x + blockIdx.x * blockDim.x;
        int y = threadIdx.y + blockIdx.y * blockDim.y;

        if (x < volume_size.x && y < volume_size.y)
        {
            int ax = x + shift.x;
            if (ax >= volume_size.x)
                ax -= volume_size.x;
            int ay = y + shift.y;
            if (ay >= volume_size.y)
                ay -= volume_size.y;

            T *pos = volume.ptr(ay) + ax;
            int z_step = volume_size.y * volume.step / sizeof(*pos);

  #pragma unroll
            for(int z = 0; z < volume_size.z; ++z)
            {
              int az = z + shift.z;
              if (az >= volume_size.z)
                az -= volume_size.z;

              float3 pt;
              pt.x = float(x);
              pt.y = float(y);
              pt.z = float(z);

              if (norm(pt - center) < radius)
              {
                if (set_to_empty)
                  pack_tsdf(1.0f, 1, *(pos + (az * z_step)));
                else
                  pack_tsdf(0.f, 0, *(pos + (az * z_step)));
              }
            }
        }
      }

      template<typename T>
      __global__ void
      clearBBoxKernel(PtrStep<T> volume,int3 volume_size,int3 shift,float3 m,float3 M,bool set_to_empty)
      {
        int x = threadIdx.x + blockIdx.x * blockDim.x;
        int y = threadIdx.y + blockIdx.y * blockDim.y;

        if (x < volume_size.x && y < volume_size.y)
        {
            int ax = x + shift.x;
            if (ax >= volume_size.x)
                ax -= volume_size.x;
            int ay = y + shift.y;
            if (ay >= volume_size.y)
                ay -= volume_size.y;

            T *pos = volume.ptr(ay) + ax;
            int z_step = volume_size.y * volume.step / sizeof(*pos);

  #pragma unroll
            for(int z = 0; z < volume_size.z; ++z)
            {
              int az = z + shift.z;
              if (az >= volume_size.z)
                az -= volume_size.z;

              float3 pt;
              pt.x = float(x);
              pt.y = float(y);
              pt.z = float(z);

              if ((pt.x >= m.x) && (pt.y >= m.y) && (pt.z >= m.z) &&
                (pt.x < M.x) && (pt.y < M.y) && (pt.z < M.z))
              {
                if (set_to_empty)
                  pack_tsdf(1.0f, 1, *(pos + (az * z_step)));
                else
                  pack_tsdf(0.f, 0, *(pos + (az * z_step)));
              }
            }
        }
      }

      template<typename T>
      __global__ void
      clearCylinderKernel(PtrStep<T> volume,int3 volume_size,int3 shift,float3 cylinder_center,float3 height_bearing,
                          float radius,float half_height,bool set_to_empty)
      {
        int x = threadIdx.x + blockIdx.x * blockDim.x;
        int y = threadIdx.y + blockIdx.y * blockDim.y;

        if (x < volume_size.x && y < volume_size.y)
        {
            int ax = x + shift.x;
            if (ax >= volume_size.x)
                ax -= volume_size.x;
            int ay = y + shift.y;
            if (ay >= volume_size.y)
                ay -= volume_size.y;

            T *pos = volume.ptr(ay) + ax;
            int z_step = volume_size.y * volume.step / sizeof(*pos);

  #pragma unroll
            for(int z = 0; z < volume_size.z; ++z)
            {
              int az = z + shift.z;
              if (az >= volume_size.z)
                az -= volume_size.z;

              float3 pt;
              pt.x = float(x);
              pt.y = float(y);
              pt.z = float(z);

              // project the point onto the cylinder height segment
              float3 projected_pt = cylinder_center - height_bearing * dot(cylinder_center - pt,height_bearing);

              if (norm(cylinder_center - projected_pt) < half_height && // check in height segment
                  norm(projected_pt - pt) < radius) // check in radius
              {
                if (set_to_empty)
                  pack_tsdf(1.0f, 1, *(pos + (az * z_step)));
                else
                  pack_tsdf(0.f, 0, *(pos + (az * z_step)));
              }
            }
        }
      }

      template<typename T>
      __global__ void
      clearSliceKernel (PtrStep<T> volume, pcl::gpu::kinfuLS::tsdf_buffer buffer, int3 minBounds, int3 maxBounds)
      {
        int x = threadIdx.x + blockIdx.x * blockDim.x;
        int y = threadIdx.y + blockIdx.y * blockDim.y;
            
        //compute relative indices
        int idX, idY;
        
        if(x < minBounds.x)
          idX = x + buffer.voxels_size.x;
        else
          idX = x;
        
        if(y < minBounds.y)
          idY = y + buffer.voxels_size.y;
        else
          idY = y;	 
                
        
        if ( x < buffer.voxels_size.x && y < buffer.voxels_size.y)
        {
            if( (idX >= minBounds.x && idX < maxBounds.x) || (idY >= minBounds.y && idY < maxBounds.y) )
            {
              // BLACK ZONE => clear on all Z values

              ///Move along z axis
              #pragma unroll
              for(int z = 0; z < buffer.voxels_size.z; ++z)
              {
                T *pos = volume.ptr(y + z * buffer.voxels_size.y) + x;
                  
                pack_tsdf (0.f, 0, *pos);
              }
            }
            else /* if( idX > maxBounds.x && idY > maxBounds.y)*/
            {
              
                ///RED ZONE  => clear only appropriate Z

              int idZ = minBounds.z;
              if (maxBounds.z < 0)
                idZ += maxBounds.z;

              if (idZ < 0)
                idZ += buffer.voxels_size.z;

              int nbSteps = abs(maxBounds.z);

              #pragma unroll
              for(int z = 0; z < nbSteps; ++z)
              {
                ///If we went outside of the memory, make sure we go back to the begining of it
                if(idZ + z >= buffer.voxels_size.z)
                  idZ -= buffer.voxels_size.z;

                T *pos = volume.ptr(y + (idZ + z) * buffer.voxels_size.y) + x;
                  
                pack_tsdf (0.f, 0, *pos);
              }
            } //else /* if( idX > maxBounds.x && idY > maxBounds.y)*/
        } // if ( x < VOLUME_X && y < VOLUME_Y)
      } // clearSliceKernel
   
      void
      initVolume (int3 voxels_size,PtrStep<short2> volume)
      {
        dim3 block (16, 16);
        dim3 grid (1, 1, 1);
        grid.x = divUp (voxels_size.x, block.x);
        grid.y = divUp (voxels_size.y, block.y);

        initializeVolume<<<grid, block>>>(voxels_size,volume);
        cudaSafeCall ( hipGetLastError () );
        cudaSafeCall (hipDeviceSynchronize ());
      }

      void
      clearSphere(PtrStep<short2> volume,const int3 voxels_size,int3 tsdf_origin,float3 center,float radius,
                  const bool set_to_empty)
      {
        dim3 block (32, 16);
        dim3 grid (1, 1, 1);
        grid.x = divUp (voxels_size.x, block.x);
        grid.y = divUp (voxels_size.y, block.y);

        clearSphereKernel<<<grid, block>>>(volume,voxels_size,tsdf_origin,center,radius,set_to_empty);
        cudaSafeCall ( hipGetLastError () );
        cudaSafeCall (hipDeviceSynchronize ());
      }

      void
      clearBBox(PtrStep<short2> volume, const int3 voxels_size, const int3& origin, const float3& m, const float3& M,
                const bool set_to_empty)
      {
        dim3 block (32, 16);
        dim3 grid (1, 1, 1);
        grid.x = divUp (voxels_size.x, block.x);
        grid.y = divUp (voxels_size.y, block.y);

        clearBBoxKernel<<<grid, block>>>(volume,voxels_size,origin,m,M,set_to_empty);
        cudaSafeCall ( hipGetLastError () );
        cudaSafeCall (hipDeviceSynchronize ());
      }

      void
      clearCylinder(PtrStep<short2> volume, const int3 voxels_size, const int3& origin,
                    float3 cylinder_center, float3 height_bearing,
                    float radius, float half_height, bool set_to_empty)
      {
        dim3 block (32, 16);
        dim3 grid (1, 1, 1);
        grid.x = divUp (voxels_size.x, block.x);
        grid.y = divUp (voxels_size.y, block.y);

        clearCylinderKernel<<<grid, block>>>(volume,voxels_size,origin,cylinder_center,height_bearing,
                                             radius,half_height,set_to_empty);
        cudaSafeCall ( hipGetLastError () );
        cudaSafeCall (hipDeviceSynchronize ());
      }
    }
  }
}


namespace pcl
{
  namespace device
  {
    namespace kinfuLS
    {
      struct Tsdf
      {
        enum
        {
          CTA_SIZE_X = 32, CTA_SIZE_Y = 8,
          MAX_WEIGHT = 1 << 7
        };

        mutable PtrStep<short2> volume;
        float3 cell_size;

        Intr intr;

        Mat33 Rcurr_inv;
        float3 tcurr;

        PtrStepSz<ushort> depth_raw; //depth in mm

        float tranc_dist_mm;

        __device__ __forceinline__ float3
        getVoxelGCoo (int x, int y, int z) const
        {
          float3 coo = make_float3 (x, y, z);
          coo += 0.5f;         //shift to cell center;

          coo.x *= cell_size.x;
          coo.y *= cell_size.y;
          coo.z *= cell_size.z;

          return coo;
        }

        __device__ __forceinline__ void
        operator () () const
        {
          int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
          int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

          if (x >= VOLUME_X || y >= VOLUME_Y)
            return;

          short2 *pos = volume.ptr (y) + x;
          int elem_step = volume.step * VOLUME_Y / sizeof(*pos);

          for (int z = 0; z < VOLUME_Z; ++z, pos += elem_step)
          {
            float3 v_g = getVoxelGCoo (x, y, z);            //3 // p

            //tranform to curr cam coo space
            float3 v = Rcurr_inv * (v_g - tcurr);           //4

            int2 coo;           //project to current cam
            coo.x = __float2int_rn (v.x * intr.fx / v.z + intr.cx);
            coo.y = __float2int_rn (v.y * intr.fy / v.z + intr.cy);

            if (v.z > 0 && coo.x >= 0 && coo.y >= 0 && coo.x < depth_raw.cols && coo.y < depth_raw.rows)           //6
            {
              int Dp = depth_raw.ptr (coo.y)[coo.x];

              if (Dp != 0)
              {
                float xl = (coo.x - intr.cx) / intr.fx;
                float yl = (coo.y - intr.cy) / intr.fy;
                float lambda_inv = rsqrtf (xl * xl + yl * yl + 1);

                float sdf = 1000 * norm (tcurr - v_g) * lambda_inv - Dp; //mm

                sdf *= (-1);

                if (sdf >= -tranc_dist_mm)
                {
                  float tsdf = fmin (1.f, sdf / tranc_dist_mm);

                  int weight_prev;
                  float tsdf_prev;

                  //read and unpack
                  unpack_tsdf (*pos, tsdf_prev, weight_prev);

                  const int Wrk = 1;

                  float tsdf_new = (tsdf_prev * weight_prev + Wrk * tsdf) / (weight_prev + Wrk);
                  int weight_new = min (weight_prev + Wrk, MAX_WEIGHT);

                  pack_tsdf (tsdf_new, weight_new, *pos);
                }
              }
            }
          }
        }
      };

      template<typename T>
      __global__ void
      uploadKnownToTSDFSliceKernel (PtrStep<T> volume, pcl::gpu::kinfuLS::tsdf_buffer buffer, int3 minBounds, int3 maxBounds,
        PtrStep<short> known_status)
      {
        int x = threadIdx.x + blockIdx.x * blockDim.x;
        int y = threadIdx.y + blockIdx.y * blockDim.y;

        //compute relative indices
        int idX, idY;

        if(x < minBounds.x)
          idX = x + buffer.voxels_size.x;
        else
          idX = x;

        if(y < minBounds.y)
          idY = y + buffer.voxels_size.y;
        else
          idY = y;


        if ( x < buffer.voxels_size.x && y < buffer.voxels_size.y)
        {
            if( (idX >= minBounds.x && idX < maxBounds.x) || (idY >= minBounds.y && idY < maxBounds.y) )
            {
                // BLACK ZONE => clear on all Z values

                ///Move along z axis
                #pragma unroll
                for(int z = 0; z < buffer.voxels_size.z; ++z)
                {
                  T *pos = volume.ptr(y + z * buffer.voxels_size.y) + x;

                  short * ks = known_status.ptr(y + z * buffer.voxels_size.y) + x;
                  const short increment = *ks;

                  if (increment) {
                    float tsdf;
                    int w;
                    unpack_tsdf(*pos, tsdf, w);
                    if (w == 0)
                      tsdf = 1.0;
                    pack_tsdf (tsdf, min(increment + w,(Tsdf::MAX_WEIGHT)), *pos);
                  }
                }
            }
            else /* if( idX > maxBounds.x && idY > maxBounds.y)*/
            {

                ///RED ZONE  => clear only appropriate Z

                int idZ = minBounds.z;
                if (maxBounds.z < 0)
                  idZ += maxBounds.z;

                if (idZ < 0)
                  idZ += buffer.voxels_size.z;

                int nbSteps = abs(maxBounds.z);

                #pragma unroll
                for(int z = 0; z < nbSteps; ++z)
                {
                  ///If we went outside of the memory, make sure we go back to the begining of it
                  if(idZ + z >= buffer.voxels_size.z)
                    idZ -= buffer.voxels_size.z;

                  T *pos = volume.ptr(y + (idZ + z) * buffer.voxels_size.y) + x;

                  short * ks = known_status.ptr(y + (idZ + z) * buffer.voxels_size.y) + x;
                  const short increment = *ks;

                  if (increment) {
                    float tsdf;
                    int w;
                    unpack_tsdf(*pos, tsdf, w);
                    if (w == 0)
                      tsdf = 1.0;
                    pack_tsdf (tsdf, min(increment + w,(Tsdf::MAX_WEIGHT)), *pos);
                  }
                }
            } //else /* if( idX > maxBounds.x && idY > maxBounds.y)*/
        } // if ( x < VOLUME_X && y < VOLUME_Y)
      } // uploadKnownToTSDFSliceKernel

      __global__ void
      integrateTsdfKernel (const Tsdf tsdf) {
        tsdf ();
      }

      __global__ void
      tsdf2 (PtrStep<short2> volume, const float tranc_dist_mm, const Mat33 Rcurr_inv, float3 tcurr,
            const Intr intr, const PtrStepSz<ushort> depth_raw, const float3 cell_size)
      {
        int x = threadIdx.x + blockIdx.x * blockDim.x;
        int y = threadIdx.y + blockIdx.y * blockDim.y;

        if (x >= VOLUME_X || y >= VOLUME_Y)
          return;

        short2 *pos = volume.ptr (y) + x;
        int elem_step = volume.step * VOLUME_Y / sizeof(short2);

        float v_g_x = (x + 0.5f) * cell_size.x - tcurr.x;
        float v_g_y = (y + 0.5f) * cell_size.y - tcurr.y;
        float v_g_z = (0 + 0.5f) * cell_size.z - tcurr.z;

        float v_x = Rcurr_inv.data[0].x * v_g_x + Rcurr_inv.data[0].y * v_g_y + Rcurr_inv.data[0].z * v_g_z;
        float v_y = Rcurr_inv.data[1].x * v_g_x + Rcurr_inv.data[1].y * v_g_y + Rcurr_inv.data[1].z * v_g_z;
        float v_z = Rcurr_inv.data[2].x * v_g_x + Rcurr_inv.data[2].y * v_g_y + Rcurr_inv.data[2].z * v_g_z;

  //#pragma unroll
        for (int z = 0; z < VOLUME_Z; ++z)
        {
          float3 vr;
          vr.x = v_g_x;
          vr.y = v_g_y;
          vr.z = (v_g_z + z * cell_size.z);

          float3 v;
          v.x = v_x + Rcurr_inv.data[0].z * z * cell_size.z;
          v.y = v_y + Rcurr_inv.data[1].z * z * cell_size.z;
          v.z = v_z + Rcurr_inv.data[2].z * z * cell_size.z;

          int2 coo;         //project to current cam
          coo.x = __float2int_rn (v.x * intr.fx / v.z + intr.cx);
          coo.y = __float2int_rn (v.y * intr.fy / v.z + intr.cy);


          if (v.z > 0 && coo.x >= 0 && coo.y >= 0 && coo.x < depth_raw.cols && coo.y < depth_raw.rows)         //6
          {
            int Dp = depth_raw.ptr (coo.y)[coo.x]; //mm

            if (Dp != 0)
            {
              float xl = (coo.x - intr.cx) / intr.fx;
              float yl = (coo.y - intr.cy) / intr.fy;
              float lambda_inv = rsqrtf (xl * xl + yl * yl + 1);

              float sdf = Dp - norm (vr) * lambda_inv * 1000; //mm


              if (sdf >= -tranc_dist_mm)
              {
                float tsdf = fmin (1.f, sdf / tranc_dist_mm);

                int weight_prev;
                float tsdf_prev;

                //read and unpack
                unpack_tsdf (*pos, tsdf_prev, weight_prev);

                const int Wrk = 1;

                float tsdf_new = (tsdf_prev * weight_prev + Wrk * tsdf) / (weight_prev + Wrk);
                int weight_new = min (weight_prev + Wrk, Tsdf::MAX_WEIGHT);

                pack_tsdf (tsdf_new, weight_new, *pos);
              }
            }
          }
          pos += elem_step;
        }       /* for(int z = 0; z < VOLUME_Z; ++z) */
      }      /* __global__ */

      //////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      void
      integrateTsdfVolume (const PtrStepSz<ushort>& depth_raw, const Intr& intr, const float3& volume_size,
                                        const Mat33& Rcurr_inv, const float3& tcurr, float tranc_dist, 
                                        PtrStep<short2> volume)
      {
        Tsdf tsdf;

        tsdf.volume = volume;  
        tsdf.cell_size.x = volume_size.x / VOLUME_X;
        tsdf.cell_size.y = volume_size.y / VOLUME_Y;
        tsdf.cell_size.z = volume_size.z / VOLUME_Z;
        
        tsdf.intr = intr;

        tsdf.Rcurr_inv = Rcurr_inv;
        tsdf.tcurr = tcurr;
        tsdf.depth_raw = depth_raw;

        tsdf.tranc_dist_mm = tranc_dist*1000; //mm

        dim3 block (Tsdf::CTA_SIZE_X, Tsdf::CTA_SIZE_Y);
        dim3 grid (divUp (VOLUME_X, block.x), divUp (VOLUME_Y, block.y));

      #if 0
        //tsdf2<<<grid, block>>>(volume, tranc_dist, Rcurr_inv, tcurr, intr, depth_raw, tsdf.cell_size);
        integrateTsdfKernel<<<grid, block>>>(tsdf);
      #endif
        cudaSafeCall ( hipGetLastError () );
        cudaSafeCall (hipDeviceSynchronize ());
      }
    }
  }
}

namespace pcl
{
  namespace device
  {
    namespace kinfuLS
    {
      __global__ void
      scaleDepth (const PtrStepSz<ushort> depth, PtrStep<float> scaled, const Intr intr)
      {
        int x = threadIdx.x + blockIdx.x * blockDim.x;
        int y = threadIdx.y + blockIdx.y * blockDim.y;

        if (x >= depth.cols || y >= depth.rows)
          return;

        int Dp = depth.ptr (y)[x];

        float xl = (x - intr.cx) / intr.fx;
        float yl = (y - intr.cy) / intr.fy;
        float lambda = sqrtf (xl * xl + yl * yl + 1);

        scaled.ptr (y)[x] = Dp * lambda/1000.f; //meters
      }

      __global__ void
      tsdf23 (const PtrStepSz<float> depthScaled, PtrStep<short2> volume,
              const float tranc_dist, const Mat33 Rcurr_inv, const float3 tcurr, const Intr intr, const float3 cell_size, const pcl::gpu::kinfuLS::tsdf_buffer buffer)
      {
        int x = threadIdx.x + blockIdx.x * blockDim.x;
        int y = threadIdx.y + blockIdx.y * blockDim.y;

        if (x >= buffer.voxels_size.x - buffer.voxels_volume_padding.x ||
            y >= buffer.voxels_size.y - buffer.voxels_volume_padding.y)
          return;
        if (x < buffer.voxels_volume_padding.x || y < buffer.voxels_volume_padding.y)
          return;

        float v_g_x = (x + 0.5f) * cell_size.x - tcurr.x;
        float v_g_y = (y + 0.5f) * cell_size.y - tcurr.y;
        float v_g_z = (0 + 0.5f) * cell_size.z - tcurr.z;

        float v_g_part_norm = v_g_x * v_g_x + v_g_y * v_g_y;

        float v_x = (Rcurr_inv.data[0].x * v_g_x + Rcurr_inv.data[0].y * v_g_y + Rcurr_inv.data[0].z * v_g_z) * intr.fx;
        float v_y = (Rcurr_inv.data[1].x * v_g_x + Rcurr_inv.data[1].y * v_g_y + Rcurr_inv.data[1].z * v_g_z) * intr.fy;
        float v_z = (Rcurr_inv.data[2].x * v_g_x + Rcurr_inv.data[2].y * v_g_y + Rcurr_inv.data[2].z * v_g_z);

        float z_scaled = 0;

        float Rcurr_inv_0_z_scaled = Rcurr_inv.data[0].z * cell_size.z * intr.fx;
        float Rcurr_inv_1_z_scaled = Rcurr_inv.data[1].z * cell_size.z * intr.fy;

        float tranc_dist_inv = 1.0f / tranc_dist;

        int idX = x + buffer.origin_GRID.x;
        if (idX >= buffer.voxels_size.x)
          idX -= buffer.voxels_size.x;

        int idY = y + buffer.origin_GRID.y;
        if (idY >= buffer.voxels_size.y)
          idY -= buffer.voxels_size.y;

  //#pragma unroll
        for (int z = buffer.voxels_volume_padding.z; z < buffer.voxels_size.z - buffer.voxels_volume_padding.z;
            ++z,
            v_g_z += cell_size.z,
            z_scaled += cell_size.z,
            v_x += Rcurr_inv_0_z_scaled,
            v_y += Rcurr_inv_1_z_scaled)
        {
          
          // As the pointer is incremented in the for loop, we have to make sure that the pointer is never outside the memory
          int idZ = z + buffer.origin_GRID.z;
          if (idZ >= buffer.voxels_size.z)
            idZ -= buffer.voxels_size.z;

          short2* pos = volume.ptr (buffer.voxels_size.y * idZ + idY) + idX;
          
          float inv_z = 1.0f / (v_z + Rcurr_inv.data[2].z * z_scaled);
          if (inv_z < 0)
              continue;

          // project to current cam
          int2 coo =
          {
            __float2int_rn (v_x * inv_z + intr.cx),
            __float2int_rn (v_y * inv_z + intr.cy)
          };

          if (coo.x >= 0 && coo.y >= 0 && coo.x < depthScaled.cols && coo.y < depthScaled.rows)         //6
          {
            float Dp_scaled = depthScaled.ptr (coo.y)[coo.x]; //meters

            float sdf = Dp_scaled - sqrtf (v_g_z * v_g_z + v_g_part_norm);

            if (Dp_scaled != 0 && sdf >= -tranc_dist) //meters
            {
              float tsdf = fmin (1.0f, sdf * tranc_dist_inv);

              //read and unpack
              float tsdf_prev;
              int weight_prev;
              unpack_tsdf (*pos, tsdf_prev, weight_prev);

              const int Wrk = 1;

              float tsdf_new = (tsdf_prev * weight_prev + Wrk * tsdf) / (weight_prev + Wrk);
              int weight_new = min (weight_prev + Wrk, Tsdf::MAX_WEIGHT);

              pack_tsdf (tsdf_new, weight_new, *pos);
            }
          }
        }       // for(int z = 0; z < VOLUME_Z; ++z)
      }      // __global__

      __global__ void
      tsdf23_only_empty (const PtrStepSz<float> depthScaled, PtrStep<short2> volume,
                         const float tranc_dist, const Mat33 Rcurr_inv, const float3 tcurr,
                         const Intr intr, const float3 cell_size, const pcl::gpu::kinfuLS::tsdf_buffer buffer)
      {
        int x = threadIdx.x + blockIdx.x * blockDim.x;
        int y = threadIdx.y + blockIdx.y * blockDim.y;

        if (x >= buffer.voxels_size.x - buffer.voxels_volume_padding.x ||
            y >= buffer.voxels_size.y - buffer.voxels_volume_padding.y)
          return;
        if (x < buffer.voxels_volume_padding.x || y < buffer.voxels_volume_padding.y)
          return;

        float v_g_x = (x + 0.5f) * cell_size.x - tcurr.x;
        float v_g_y = (y + 0.5f) * cell_size.y - tcurr.y;
        float v_g_z = (0 + 0.5f) * cell_size.z - tcurr.z;

        float v_g_part_norm = v_g_x * v_g_x + v_g_y * v_g_y;

        float v_x = (Rcurr_inv.data[0].x * v_g_x + Rcurr_inv.data[0].y * v_g_y + Rcurr_inv.data[0].z * v_g_z) * intr.fx;
        float v_y = (Rcurr_inv.data[1].x * v_g_x + Rcurr_inv.data[1].y * v_g_y + Rcurr_inv.data[1].z * v_g_z) * intr.fy;
        float v_z = (Rcurr_inv.data[2].x * v_g_x + Rcurr_inv.data[2].y * v_g_y + Rcurr_inv.data[2].z * v_g_z);

        float z_scaled = 0;

        float Rcurr_inv_0_z_scaled = Rcurr_inv.data[0].z * cell_size.z * intr.fx;
        float Rcurr_inv_1_z_scaled = Rcurr_inv.data[1].z * cell_size.z * intr.fy;

        int idX = x + buffer.origin_GRID.x;
        if (idX >= buffer.voxels_size.x)
          idX -= buffer.voxels_size.x;

        int idY = y + buffer.origin_GRID.y;
        if (idY >= buffer.voxels_size.y)
          idY -= buffer.voxels_size.y;

  //#pragma unroll
        for (int z = buffer.voxels_volume_padding.z; z < buffer.voxels_size.z - buffer.voxels_volume_padding.z;
            ++z,
            v_g_z += cell_size.z,
            z_scaled += cell_size.z,
            v_x += Rcurr_inv_0_z_scaled,
            v_y += Rcurr_inv_1_z_scaled)
        {

          // As the pointer is incremented in the for loop, we have to make sure that the pointer is never outside the memory
          int idZ = z + buffer.origin_GRID.z;
          if (idZ >= buffer.voxels_size.z)
            idZ -= buffer.voxels_size.z;

          short2* pos = volume.ptr (buffer.voxels_size.y * idZ + idY) + idX;

          float inv_z = 1.0f / (v_z + Rcurr_inv.data[2].z * z_scaled);
          if (inv_z < 0)
              continue;

          // project to current cam
          int2 coo =
          {
            __float2int_rn (v_x * inv_z + intr.cx),
            __float2int_rn (v_y * inv_z + intr.cy)
          };

          if (coo.x >= 0 && coo.y >= 0 && coo.x < depthScaled.cols && coo.y < depthScaled.rows)         //6
          {
            float Dp_scaled = depthScaled.ptr (coo.y)[coo.x]; //meters

            float sdf = Dp_scaled - sqrtf (v_g_z * v_g_z + v_g_part_norm);

            if (Dp_scaled != 0 && sdf >= tranc_dist) //meters
            {
              float tsdf = 1.0f;

              //read and unpack
              float tsdf_prev;
              int weight_prev;
              unpack_tsdf (*pos, tsdf_prev, weight_prev);

              const int Wrk = 1;

              float tsdf_new = (tsdf_prev * weight_prev + Wrk * tsdf) / (weight_prev + Wrk);
              int weight_new = min (weight_prev + Wrk, Tsdf::MAX_WEIGHT);

              pack_tsdf (tsdf_new, weight_new, *pos);
            }
          }
        }       // for(int z = 0; z < VOLUME_Z; ++z)
      }      // __global__

      __global__ void
      tsdf23normal_hack (const PtrStepSz<float> depthScaled, PtrStep<short2> volume,
                    const float tranc_dist, const Mat33 Rcurr_inv, const float3 tcurr, const Intr intr, const float3 cell_size)
      {
          int x = threadIdx.x + blockIdx.x * blockDim.x;
          int y = threadIdx.y + blockIdx.y * blockDim.y;

          if (x >= VOLUME_X || y >= VOLUME_Y)
              return;

          const float v_g_x = (x + 0.5f) * cell_size.x - tcurr.x;
          const float v_g_y = (y + 0.5f) * cell_size.y - tcurr.y;
          float v_g_z = (0 + 0.5f) * cell_size.z - tcurr.z;

          float v_g_part_norm = v_g_x * v_g_x + v_g_y * v_g_y;

          float v_x = (Rcurr_inv.data[0].x * v_g_x + Rcurr_inv.data[0].y * v_g_y + Rcurr_inv.data[0].z * v_g_z) * intr.fx;
          float v_y = (Rcurr_inv.data[1].x * v_g_x + Rcurr_inv.data[1].y * v_g_y + Rcurr_inv.data[1].z * v_g_z) * intr.fy;
          float v_z = (Rcurr_inv.data[2].x * v_g_x + Rcurr_inv.data[2].y * v_g_y + Rcurr_inv.data[2].z * v_g_z);

          float z_scaled = 0;

          float Rcurr_inv_0_z_scaled = Rcurr_inv.data[0].z * cell_size.z * intr.fx;
          float Rcurr_inv_1_z_scaled = Rcurr_inv.data[1].z * cell_size.z * intr.fy;

          float tranc_dist_inv = 1.0f / tranc_dist;

          short2* pos = volume.ptr (y) + x;
          int elem_step = volume.step * VOLUME_Y / sizeof(short2);

          //#pragma unroll
          for (int z = 0; z < VOLUME_Z;
              ++z,
              v_g_z += cell_size.z,
              z_scaled += cell_size.z,
              v_x += Rcurr_inv_0_z_scaled,
              v_y += Rcurr_inv_1_z_scaled,
              pos += elem_step)
          {
              float inv_z = 1.0f / (v_z + Rcurr_inv.data[2].z * z_scaled);
              if (inv_z < 0)
                  continue;

              // project to current cam
              int2 coo =
              {
                  __float2int_rn (v_x * inv_z + intr.cx),
                  __float2int_rn (v_y * inv_z + intr.cy)
              };

              if (coo.x >= 0 && coo.y >= 0 && coo.x < depthScaled.cols && coo.y < depthScaled.rows)         //6
              {
                  float Dp_scaled = depthScaled.ptr (coo.y)[coo.x]; //meters

                  float sdf = Dp_scaled - sqrtf (v_g_z * v_g_z + v_g_part_norm);

                  if (Dp_scaled != 0 && sdf >= -tranc_dist) //meters
                  {
                      float tsdf = fmin (1.0f, sdf * tranc_dist_inv);                                              

                      bool integrate = true;
                      if ((x > 0 &&  x < VOLUME_X-2) && (y > 0 && y < VOLUME_Y-2) && (z > 0 && z < VOLUME_Z-2))
                      {
                          const float qnan = numeric_limits<float>::quiet_NaN();
                          float3 normal = make_float3(qnan, qnan, qnan);

                          float Fn, Fp;
                          int Wn = 0, Wp = 0;
                          unpack_tsdf (*(pos + elem_step), Fn, Wn);
                          unpack_tsdf (*(pos - elem_step), Fp, Wp);

                          if (Wn > 16 && Wp > 16) 
                              normal.z = (Fn - Fp)/cell_size.z;

                          unpack_tsdf (*(pos + volume.step/sizeof(short2) ), Fn, Wn);
                          unpack_tsdf (*(pos - volume.step/sizeof(short2) ), Fp, Wp);

                          if (Wn > 16 && Wp > 16) 
                              normal.y = (Fn - Fp)/cell_size.y;

                          unpack_tsdf (*(pos + 1), Fn, Wn);
                          unpack_tsdf (*(pos - 1), Fp, Wp);

                          if (Wn > 16 && Wp > 16) 
                              normal.x = (Fn - Fp)/cell_size.x;

                          if (normal.x != qnan && normal.y != qnan && normal.z != qnan)
                          {
                              float norm2 = dot(normal, normal);
                              if (norm2 >= 1e-10)
                              {
                                  normal *= rsqrt(norm2);

                                  float nt = v_g_x * normal.x + v_g_y * normal.y + v_g_z * normal.z;
                                  float cosine = nt * rsqrt(v_g_x * v_g_x + v_g_y * v_g_y + v_g_z * v_g_z);

                                  if (cosine < 0.5)
                                      integrate = false;
                              }
                          }
                      }

                      if (integrate)
                      {
                          //read and unpack
                          float tsdf_prev;
                          int weight_prev;
                          unpack_tsdf (*pos, tsdf_prev, weight_prev);

                          const int Wrk = 1;

                          float tsdf_new = (tsdf_prev * weight_prev + Wrk * tsdf) / (weight_prev + Wrk);
                          int weight_new = min (weight_prev + Wrk, Tsdf::MAX_WEIGHT);

                          pack_tsdf (tsdf_new, weight_new, *pos);
                      }
                  }
              }
          }       // for(int z = 0; z < VOLUME_Z; ++z)
      }      // __global__

      //////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      void
      integrateTsdfVolume (const PtrStepSz<ushort>& depth, const Intr& intr,
                                        const float3& volume_size, const Mat33& Rcurr_inv, const float3& tcurr, 
                                        float tranc_dist,
                                        PtrStep<short2> volume, const pcl::gpu::kinfuLS::tsdf_buffer* buffer, DeviceArray2D<float>& depthScaled)
      {
        depthScaled.create (depth.rows, depth.cols);

        dim3 block_scale (32, 8);
        dim3 grid_scale (divUp (depth.cols, block_scale.x), divUp (depth.rows, block_scale.y));

        //scales depth along ray and converts mm -> meters. 
        scaleDepth<<<grid_scale, block_scale>>>(depth, depthScaled, intr);
        cudaSafeCall ( hipGetLastError () );

        float3 cell_size;
        cell_size.x = volume_size.x / buffer->voxels_size.x;
        cell_size.y = volume_size.y / buffer->voxels_size.y;
        cell_size.z = volume_size.z / buffer->voxels_size.z;

        //dim3 block(Tsdf::CTA_SIZE_X, Tsdf::CTA_SIZE_Y);
        dim3 block (16, 16);
        dim3 grid (divUp (buffer->voxels_size.x, block.x), divUp (buffer->voxels_size.y, block.y));

        tsdf23<<<grid, block>>>(depthScaled, volume, tranc_dist, Rcurr_inv, tcurr, intr, cell_size, *buffer);    
        //tsdf23normal_hack<<<grid, block>>>(depthScaled, volume, tranc_dist, Rcurr_inv, tcurr, intr, cell_size);

        cudaSafeCall ( hipGetLastError () );
        cudaSafeCall (hipDeviceSynchronize ());
      }

      //////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      void
      integrateTsdfVolumeOnlyEmpty (const PtrStepSz<ushort>& depth, const Intr& intr,
                                    const float3& volume_size, const Mat33& Rcurr_inv, const float3& tcurr,
                                    float tranc_dist,
                                    PtrStep<short2> volume, const pcl::gpu::kinfuLS::tsdf_buffer* buffer,
                                    DeviceArray2D<float>& depthScaled)
      {
        depthScaled.create (depth.rows, depth.cols);

        dim3 block_scale (32, 8);
        dim3 grid_scale (divUp (depth.cols, block_scale.x), divUp (depth.rows, block_scale.y));

        //scales depth along ray and converts mm -> meters.
        scaleDepth<<<grid_scale, block_scale>>>(depth, depthScaled, intr);
        cudaSafeCall ( hipGetLastError () );

        float3 cell_size;
        cell_size.x = volume_size.x / buffer->voxels_size.x;
        cell_size.y = volume_size.y / buffer->voxels_size.y;
        cell_size.z = volume_size.z / buffer->voxels_size.z;

        //dim3 block(Tsdf::CTA_SIZE_X, Tsdf::CTA_SIZE_Y);
        dim3 block (16, 16);
        dim3 grid (divUp (buffer->voxels_size.x, block.x), divUp (buffer->voxels_size.y, block.y));

        tsdf23_only_empty<<<grid, block>>>(depthScaled, volume, tranc_dist, Rcurr_inv, tcurr, intr, cell_size, *buffer);
        //tsdf23normal_hack<<<grid, block>>>(depthScaled, volume, tranc_dist, Rcurr_inv, tcurr, intr, cell_size);

        cudaSafeCall ( hipGetLastError () );
        cudaSafeCall (hipDeviceSynchronize ());
      }

      /////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      void 
      clearTSDFSlice (PtrStep<short2> volume, pcl::gpu::kinfuLS::tsdf_buffer* buffer, int shiftX, int shiftY, int shiftZ)
      {
        int newX = buffer->origin_GRID.x + shiftX;
        int newY = buffer->origin_GRID.y + shiftY;

        int3 minBounds, maxBounds;
        
        //X
        if(newX >= 0)
        {
          minBounds.x = buffer->origin_GRID.x;
          maxBounds.x = newX;
        }
        else
        {
          minBounds.x = newX + buffer->voxels_size.x;
          maxBounds.x = buffer->origin_GRID.x + buffer->voxels_size.x;
        }
        
        if(minBounds.x > maxBounds.x)
          std::swap(minBounds.x, maxBounds.x);
      
        //Y
        if(newY >= 0)
        {
          minBounds.y = buffer->origin_GRID.y;
          maxBounds.y = newY;
        }
        else
        {
          minBounds.y = newY + buffer->voxels_size.y;
          maxBounds.y = buffer->origin_GRID.y + buffer->voxels_size.y;
        }
        
        if(minBounds.y > maxBounds.y)
          std::swap(minBounds.y, maxBounds.y);
        
        //Z
        minBounds.z = buffer->origin_GRID.z;
        maxBounds.z = shiftZ;
      
        // call kernel
        dim3 block (32, 16);
        dim3 grid (1, 1, 1);
        grid.x = divUp (buffer->voxels_size.x, block.x);      
        grid.y = divUp (buffer->voxels_size.y, block.y);
        
        clearSliceKernel<<<grid, block>>>(volume, *buffer, minBounds, maxBounds);
        cudaSafeCall ( hipGetLastError () );
        cudaSafeCall (hipDeviceSynchronize ());        
      }

    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      void
      uploadKnownToTSDFSlice (PtrStep<short2> volume, pcl::gpu::kinfuLS::tsdf_buffer* buffer, int shiftX, int shiftY, int shiftZ,
        PtrStep<short> known_status)
      {
        int oldX = buffer->origin_GRID.x - shiftX;
        int oldY = buffer->origin_GRID.y - shiftY;
        int oldZ = buffer->origin_GRID.z - shiftZ;

        int3 minBounds, maxBounds;

        //X
        if(oldX >= 0)
        {
          minBounds.x = buffer->origin_GRID.x;
          maxBounds.x = oldX;
        }
        else
        {
          minBounds.x = oldX + buffer->voxels_size.x;
          maxBounds.x = buffer->origin_GRID.x + buffer->voxels_size.x;
        }

        if(minBounds.x > maxBounds.x)
          std::swap(minBounds.x, maxBounds.x);


        //Y
        if(oldY >= 0)
        {
          minBounds.y = buffer->origin_GRID.y;
          maxBounds.y = oldY;
        }
        else
        {
          minBounds.y = oldY + buffer->voxels_size.y;
          maxBounds.y = buffer->origin_GRID.y + buffer->voxels_size.y;
        }

        if(minBounds.y > maxBounds.y)
          std::swap(minBounds.y, maxBounds.y);

        while (oldZ < 0)
          oldZ += buffer->voxels_size.z;
        while (oldZ >= buffer->voxels_size.z)
          oldZ -= buffer->voxels_size.z;

        //Z
        minBounds.z = oldZ;
        maxBounds.z = shiftZ;

        // call kernel
        dim3 block (32, 16);
        dim3 grid (1, 1, 1);
        grid.x = divUp (buffer->voxels_size.x, block.x);
        grid.y = divUp (buffer->voxels_size.y, block.y);

        uploadKnownToTSDFSliceKernel<<<grid, block>>>(volume, *buffer, minBounds, maxBounds, known_status);
        cudaSafeCall ( hipGetLastError () );
        cudaSafeCall (hipDeviceSynchronize ());
      }

      template<typename T>
      __global__ void
      uploadKnownToBBoxKernel(PtrStep<T> volume,int3 volume_size,int3 shift,int3 m,int3 M,
                              PtrStep<short> known_status)
      {
        int x = threadIdx.x + blockIdx.x * blockDim.x;
        int y = threadIdx.y + blockIdx.y * blockDim.y;

        int3 known_status_size;
        known_status_size.x = M.x - m.x;
        known_status_size.y = M.y - m.y;
        known_status_size.z = M.z - m.z;

        if (x < volume_size.x && y < volume_size.y)
        {
            int ax = x + shift.x;
            if (ax >= volume_size.x)
                ax -= volume_size.x;
            int ay = y + shift.y;
            if (ay >= volume_size.y)
                ay -= volume_size.y;

            T *pos = volume.ptr(ay) + ax;

  #pragma unroll
            for(int z = 0; z < volume_size.z; ++z)
            {
              int az = z + shift.z;
              if (az >= volume_size.z)
                az -= volume_size.z;

              float3 pt;
              pt.x = float(x);
              pt.y = float(y);
              pt.z = float(z);

              if ((pt.x >= m.x) && (pt.y >= m.y) && (pt.z >= m.z) &&
                (pt.x < M.x) && (pt.y < M.y) && (pt.z < M.z))
              {
                short * ks = known_status.ptr((y - m.y) + (z - m.z) * known_status_size.y) + (x - m.x);
                const short increment = *ks;

                if (increment) {
                  float tsdf;
                  int w;
                  unpack_tsdf(*pos, tsdf, w);
                  if (w == 0)
                    tsdf = 1.0;
                  pack_tsdf (tsdf, min(increment + w,(Tsdf::MAX_WEIGHT)), *pos);
                }
              }
            }
        }
      }

      void
      uploadKnownToBBox (PtrStep<short2> volume, const int3 voxels_size,const int3& origin,
                         const int3& m,const int3& M,
                         PtrStep<short> known_status)
      {
        dim3 block (32, 16);
        dim3 grid (1, 1, 1);
        grid.x = divUp (voxels_size.x, block.x);
        grid.y = divUp (voxels_size.y, block.y);

        uploadKnownToBBoxKernel<<<grid, block>>>(volume,voxels_size,origin,m,M,known_status);
        cudaSafeCall ( hipGetLastError () );
        cudaSafeCall (hipDeviceSynchronize ());
      }
    }
  }
}
